#include "hip/hip_runtime.h"
/*
Author: Benjamin Kubwimana
Class: ECE4122 or ECE6122 (section Q)
Last Date Modified: 11/07/2024
Description:
cuda kernel implementations for grid updates and neighbor status computation
*/

// cuda_kernels.cu
#include "cuda_kernels.cuh"
#include <hip/hip_runtime.h>
#include <iostream>
#include ""
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <ctime>
#include <vector>
#include <chrono>

using namespace std;

// Kernel function to update the grid
__global__ void updateGridKernel(uint8_t* grid, uint8_t* newGrid, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    int y = blockIdx.y * blockDim.y + threadIdx.y; 
    if (x < width && y < height)
    {
        int idx = y * width + x;
        int neighbors = countNeighbors(grid, x, y, width, height);
        if (grid[idx])
        {
            // Alive cell
            if (neighbors < 2 || neighbors > 3)
            {
                newGrid[idx] = 0; // Cell dies
            }
            else
            {
                //stays alive
                newGrid[idx] = 1; // 
            }
        }
        else
        {
            // Dead cell
            if (neighbors == 3)
            {   //becomes alive
                newGrid[idx] = 1;
            }
            else
            {   //stays dead
                newGrid[idx] = 0;
            }
        }
    }
}

// function to count neighbors on GPU Device
__device__ int countNeighbors(uint8_t* grid, int x, int y, int width, int height) {
    int count = 0;
    for (int dy = -1; dy <= 1; ++dy)
    {
        for (int dx = -1; dx <= 1; ++dx)
        {
            if (dx == 0 && dy == 0) continue;
            int nx = x + dx;
            int ny = y + dy;
            if (nx >= 0 && nx < width && ny >= 0 && ny < height)
            {
                int idx = ny * width + nx;
                count += grid[idx];
            }
        }
    }
    return count;
}


// Function to update the grid 
void updateGrid(uint8_t* grid, uint8_t* newGrid, uint8_t* d_grid, uint8_t* d_newGrid, MemType memory, int threadsPerBlock, int width, int height)
{
    size_t size = width * height * sizeof(uint8_t);

    hipError_t err;

    if (memory == MemType::NORMAL || memory == MemType::PINNED) {
        // data from host to device
        err = hipMemcpy(d_grid, grid, size, hipMemcpyHostToDevice);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy data from host to device for d_grid (error code %s)!\n", hipGetErrorString(err));
            hipFree(d_grid);
            hipFree(d_newGrid);
            exit(EXIT_FAILURE);
        }

        //block and grid dimensions
        dim3 blockDim(threadsPerBlock, 1);
        dim3 gridDim(
            (width + blockDim.x - 1) / blockDim.x,
            (height + blockDim.y - 1) / blockDim.y
        );

        // kernel launch
        updateGridKernel << <gridDim, blockDim >> > (d_grid, d_newGrid, width, height);
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch updateGridKernel (error code %s)!\n", hipGetErrorString(err));
            hipFree(d_grid);
            hipFree(d_newGrid);
            exit(EXIT_FAILURE);
        }

        // Wait for GPU to finish
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to synchronize device (error code %s)!\n", hipGetErrorString(err));
            hipFree(d_grid);
            hipFree(d_newGrid);
            exit(EXIT_FAILURE);
        }

        // result back to host
        err = hipMemcpy(newGrid, d_newGrid, size, hipMemcpyDeviceToHost);
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to copy data from device to host for newGrid (error code %s)!\n", hipGetErrorString(err));
            hipFree(d_grid);
            hipFree(d_newGrid);
            exit(EXIT_FAILURE);
        }
    }
    else if (memory == MemType::MANAGED) {

        // block and grid dimensions
        dim3 blockDim(threadsPerBlock, 1);
        dim3 gridDim(
            (width + blockDim.x - 1) / blockDim.x,
            (height + blockDim.y - 1) / blockDim.y
        );

        // kernel launch
        updateGridKernel << <gridDim, blockDim >> > (grid, newGrid, width, height);
        err = hipGetLastError();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to launch updateGridKernel (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }

        // Wait for GPU to finish
        err = hipDeviceSynchronize();
        if (err != hipSuccess) {
            fprintf(stderr, "Failed to synchronize device (error code %s)!\n", hipGetErrorString(err));
            exit(EXIT_FAILURE);
        }
    }
}
